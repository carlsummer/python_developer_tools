#include "hip/hip_runtime.h"

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cfloat>

#include <iostream>

using std::cout;
using std::endl;

#define BLOCKSIZE 512


template<typename scalar_t>
__global__ void compute_numer_denor(const int nthreads,
                            const scalar_t *logits,
                            const int64_t *labels,
                            scalar_t *numer,
                            scalar_t *denor,
                            const float p, const float smooth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    int batch_size = gridDim.y;
    int sample_idx = blockIdx.y;
    int sample_size = nthreads / batch_size;
/* 
 *     Tips about shared memory:
 *     1. torch will instantiate the template with three types: double, float, half;
 *     2. these three types should not share same definitions of shared memory;
 *     3. so one method is to use static shared memory with memory size explicitly assigned, and another method is to allocate shared memory with same raw type, such as unsigned char here, and then cast the pointer according to different template types
 *  */
    // method1: use static sized shared memory
    // __shared__ scalar_t sdata[BLOCKSIZE * 2];
    // method2: allocate with raw uchar type and then cast in different kernel
    extern __shared__ __align__(sizeof(scalar_t)) unsigned char sdata_raw[];
    scalar_t *sdata = reinterpret_cast<scalar_t*>(sdata_raw);

    sdata[threadIdx.x] = 0; // numer
    sdata[threadIdx.x + blockDim.x] = 0; // denor
    __syncthreads();

    for (int i{tid}; i < sample_size; i+=stride) {
        int idx = sample_idx * sample_size + i;
        scalar_t prob = 1. / (1. + expf(-logits[idx]));
        scalar_t lb = (scalar_t)labels[idx];

        sdata[threadIdx.x] += 2 * prob * lb;
        sdata[threadIdx.x + blockDim.x] += powf(prob, p) + lb;
    }
    __syncthreads();

    for (int s=1; s < blockDim.x; s*=2) {
        int idx = 2 * s * threadIdx.x;
        if (idx < blockDim.x && idx + s < blockDim.x) {
            sdata[idx] += sdata[idx + s];
        }
        idx += blockDim.x;
        if (idx < (blockDim.x + blockDim.x) && idx + s < (blockDim.x + blockDim.x)) {
            sdata[idx] += sdata[idx + s];
        }
        __syncthreads();
    }

    if (blockIdx.x == 0 && threadIdx.x == 0) {
        sdata[0] += smooth;
        sdata[blockDim.x] += smooth;
    }
    if (threadIdx.x == 0) {
        atomicAdd(&numer[sample_idx], sdata[0]);
        atomicAdd(&denor[sample_idx], sdata[blockDim.x]);
    }
}

// kernel function for forward and backward
template<typename scalar_t>
__global__ void SoftDiceForward(const int nthreads,
                            scalar_t *loss,
                            const scalar_t *numer,
                            const scalar_t *denor) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    for (int i{tid}; i < nthreads; i+=stride) {
        loss[i] = 1. - numer[i] / denor[i];
    }
}


template<typename scalar_t>
__global__ void SoftDiceBackward(const int nthreads,
                             const scalar_t *logits,
                             const int64_t *labels,
                             const scalar_t *grad,
                             const scalar_t *numer,
                             const scalar_t *denor,
                             scalar_t *grad_logits,
                             const float p, const float smooth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int sample_idx = blockIdx.y;
    int stride = gridDim.x * blockDim.x;
    int batch_size = gridDim.y;
    int sample_size = nthreads / batch_size;

    scalar_t numer_val = numer[sample_idx];
    scalar_t denor_val = denor[sample_idx];
    scalar_t grad_val = grad[sample_idx];
    for (int i{tid}; i < sample_size; i+=stride) {
        int idx = sample_idx * sample_size + i;
        scalar_t prob = 1. / (1. + expf(-logits[idx]));
        scalar_t lb = (scalar_t)labels[idx];
        scalar_t m = numer_val - 2. * (prob * lb);
        scalar_t n = denor_val - powf(prob, p);
        scalar_t g = -powf(prob, p - 1.) * p * m;
        if (lb == 1) {
            g += powf(prob, p) * 2. * (1. - p) + (n * 2.);
        }
        g = - (g / powf(powf(prob, p) + n, 2.)) * prob * (1. - prob);
        grad_logits[idx] = grad_val * g;
    }
}


// cuda forward and backward
at::Tensor SoftDice_forward_cuda(const at::Tensor &logits,
                                  const at::Tensor &labels,
                                  const float p,
                                  const float smooth) {
    // CHECK type and shape
    AT_ASSERTM(logits.type().is_cuda(), "logits should be cuda");
    AT_ASSERTM(labels.type().is_cuda(), "labels should be cuda");

    const int batchsize = logits.size(0);
    const int num_samples = logits.numel();
    const int sample_size = num_samples / batchsize;
    // allocate memory and cuda grid/block
    auto numer = torch::zeros({batchsize}, logits.options());
    auto denor = torch::zeros({batchsize}, logits.options());
    auto losses = torch::empty({batchsize}, logits.options());
    // Note: should use torch::zeros rather than at::zeros, torch::zeros is variable
    // and at::zeros is tensor

    dim3 grid1(std::min(
        THCCeilDiv((int64_t)sample_size, (int64_t)BLOCKSIZE), (int64_t)4096
    ), batchsize);
    dim3 block1(BLOCKSIZE);
    dim3 grid2(1);
    dim3 block2(BLOCKSIZE);
    if (losses.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return losses;
    }

    // call kernel
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(losses.scalar_type(), "soft dice forward", [&] {
        int shm_size = BLOCKSIZE * sizeof(scalar_t) * 2;
        compute_numer_denor<scalar_t><<<grid1, block1, shm_size, at::cuda::getCurrentCUDAStream()>>>(
            num_samples, 
            logits.contiguous().data<scalar_t>(), 
            labels.contiguous().data<int64_t>(), 
            numer.contiguous().data<scalar_t>(),
            denor.contiguous().data<scalar_t>(),
            p, smooth
        );
        SoftDiceForward<scalar_t><<<grid2, block2, 0, at::cuda::getCurrentCUDAStream()>>>(
            batchsize,
            losses.contiguous().data<scalar_t>(),
            numer.contiguous().data<scalar_t>(),
            denor.contiguous().data<scalar_t>()
        );
    });
    THCudaCheck(hipGetLastError());
    return losses;
}


at::Tensor SoftDice_backward_cuda(const at::Tensor &grad,
                                  const at::Tensor &logits,
                                  const at::Tensor &labels,
                                  const float p,
                                  const float smooth) {
    // CHECK type and shape
    AT_ASSERTM(grad.type().is_cuda(), "grad should be cuda");
    AT_ASSERTM(logits.type().is_cuda(), "logits should be cuda");
    AT_ASSERTM(labels.type().is_cuda(), "labels should be cuda");

    const int batchsize = logits.size(0);
    const int num_samples = logits.numel();
    const int sample_size = num_samples / batchsize;
    // allocate memory and cuda grid/block
    auto grad_logits = torch::empty_like(logits);
    auto numer = torch::zeros({batchsize}, logits.options());
    auto denor = torch::zeros({batchsize}, logits.options());

    dim3 grid(std::min(
        THCCeilDiv((int64_t)sample_size, (int64_t)BLOCKSIZE), (int64_t)4096
    ), batchsize);
    dim3 block(BLOCKSIZE);
    if (grad_logits.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return grad_logits;
    }

    // call kernel
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_logits.scalar_type(), "soft dice backwrd", [&] {
        int shm_size = BLOCKSIZE * sizeof(scalar_t) * 2;
        compute_numer_denor<scalar_t><<<grid, block, shm_size, at::cuda::getCurrentCUDAStream()>>>(
            num_samples, 
            logits.contiguous().data<scalar_t>(), 
            labels.contiguous().data<int64_t>(), 
            numer.contiguous().data<scalar_t>(),
            denor.contiguous().data<scalar_t>(),
            p, smooth
        );
        SoftDiceBackward<scalar_t><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            num_samples, 
            logits.contiguous().data<scalar_t>(), 
            labels.contiguous().data<int64_t>(),
            grad.contiguous().data<scalar_t>(),
            numer.contiguous().data<scalar_t>(),
            denor.contiguous().data<scalar_t>(),
            grad_logits.contiguous().data<scalar_t>(),
            p, smooth
        );
    });
    THCudaCheck(hipGetLastError());
    return grad_logits;
}

// python inferface
at::Tensor SoftDice_forward(const at::Tensor &logits,
                             const at::Tensor &labels,
                             const float p,
                             const float smooth) {
    if (!(logits.type().is_cuda() && labels.type().is_cuda())) {
        AT_ERROR("this dice loss only supports gpu mode\n");
    } 
    at::DeviceGuard guard(logits.device());
    return SoftDice_forward_cuda(logits, labels, p, smooth);
}

at::Tensor SoftDice_backward(const at::Tensor &grad,
                                  const at::Tensor &logits,
                                  const at::Tensor &labels,
                                  const float p,
                                  const float smooth) {
    // TODO: try AT_ASSERTM
    if (!(logits.type().is_cuda() && labels.type().is_cuda())) {
        AT_ERROR("this dice loss only supports gpu mode\n");
    } 
    at::DeviceGuard guard(logits.device());
    return SoftDice_backward_cuda(grad, logits, labels, p, smooth);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("soft_dice_forward", &SoftDice_forward, "soft-dice forward");
    m.def("soft_dice_backward", &SoftDice_backward, "soft-dice backward");
}
