#include "hip/hip_runtime.h"

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cfloat>

#include "common.hpp"

using math_ops::Exp;
using math_ops::Log;
using math_ops::Log1p;
using math_ops::Pow;



template<typename scalar_t>
__global__ void FocalLossForward(const int nthreads,
                                 const scalar_t *logits,
                                 const int64_t *labels,
                                 scalar_t *loss,
                                 const scalar_t gamma, const scalar_t alpha) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    const scalar_t one(1.);
    for (int i{tid}; i < nthreads; i+=stride) {
        scalar_t lgt = logits[i];
        scalar_t prob = one / (one + Exp(-lgt));
        scalar_t log_p, log_1_p;
        if (lgt >= 0) {
            // log_p = -Log(one + Exp(-lgt));
            log_p = -Log1p(Exp(-lgt));
            log_1_p = -lgt + log_p;
        } else {
            // log_1_p = -Log(one + Exp(lgt));
            log_1_p = -Log1p(Exp(lgt));
            log_p = lgt + log_1_p;
        }
        scalar_t term1 = Pow(one - prob, gamma) * log_p;
        scalar_t term2 = Pow(prob, gamma) * log_1_p;
        loss[i] = -alpha * term1 * labels[i] - (one - alpha) * term2 * (one - labels[i]);
    }
}

template<typename scalar_t>
__global__ void FocalLossBackward(const int nthreads,
                                  const scalar_t *logits,
                                  const int64_t *labels,
                                  const scalar_t *grad_loss,
                                  scalar_t *grad_logits,
                                  const scalar_t gamma, const scalar_t alpha) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    const scalar_t one(1.);
    for (int i{tid}; i < nthreads; i+=stride) {
        scalar_t lgt = logits[i];
        scalar_t prob = one / (one + Exp(-lgt));
        scalar_t log_p, log_1_p;
        if (lgt >= 0) {
            // log_p = -Log(one + Exp(-lgt));
            log_p = -Log1p(Exp(-lgt));
            log_1_p = -lgt + log_p;
        } else {
            // log_1_p = -Log(one + Exp(lgt));
            log_1_p = -Log1p(Exp(lgt));
            log_p = lgt + log_1_p;
        }
        scalar_t term1 = Pow(one - prob, gamma) * (one - prob - gamma * prob * log_p);
        scalar_t term2 = Pow(prob, gamma) * (gamma * (one - prob) * log_1_p - prob);
        grad_logits[i] = -alpha * term1 * labels[i] - (one - alpha) * term2 * (one - labels[i]);
        grad_logits[i] = grad_logits[i] * grad_loss[i];
    }
}


at::Tensor FocalLoss_forward_cuda(const at::Tensor &logits,
                                  const at::Tensor &labels,
                                  const float gamma,
                                  const float alpha) {
    // CHECK type and shape
    AT_ASSERTM(logits.device().type() == c10::kCUDA, "logits should be cuda");
    AT_ASSERTM(labels.device().type() == c10::kCUDA, "labels should be cuda");

    // allocate memory and cuda grid/block
    auto losses = at::empty_like(logits);

    const int num_samples = logits.numel();
    dim3 grid(std::min(
        THCCeilDiv((int64_t)num_samples, (int64_t)512), (int64_t)4096
    ));
    dim3 block(512);
    if (losses.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return losses;
    }

    // call kernel
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(logits.scalar_type(), "focal forward", [&] {
        FocalLossForward<scalar_t><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            num_samples, 
            logits.contiguous().data_ptr<scalar_t>(), 
            labels.contiguous().data_ptr<int64_t>(),
            losses.contiguous().data_ptr<scalar_t>(),
            scalar_t(gamma), scalar_t(alpha)
        );
    });
    THCudaCheck(hipGetLastError());
    return losses;
}


at::Tensor FocalLoss_backward_cuda(const at::Tensor &grad,
                                  const at::Tensor &logits,
                                  const at::Tensor &labels,
                                  const float gamma,
                                  const float alpha) {
    // CHECK type and shape
    AT_ASSERTM(logits.device().type() == c10::kCUDA, "logits should be cuda");
    AT_ASSERTM(labels.device().type() == c10::kCUDA, "labels should be cuda");

    /* allocate memory and cuda grid/block */
    auto grad_logits = at::empty_like(logits);
    const int num_samples = logits.numel();
    dim3 grid(std::min(
        THCCeilDiv((int64_t)num_samples, (int64_t)512), (int64_t)4096
    ));
    dim3 block(512);
    if (grad_logits.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return grad_logits;
    }

    // call kernel
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(logits.scalar_type(), "focal backwrd", [&] {
        FocalLossBackward<scalar_t><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            num_samples, 
            logits.contiguous().data_ptr<scalar_t>(), 
            labels.contiguous().data_ptr<int64_t>(),
            grad.contiguous().data_ptr<scalar_t>(),
            grad_logits.contiguous().data_ptr<scalar_t>(),
            scalar_t(gamma), scalar_t(alpha)
        );
    });
    THCudaCheck(hipGetLastError());
    return grad_logits;
}

// python inferface
at::Tensor FocalLoss_forward(const at::Tensor &logits,
                             const at::Tensor &labels,
                             const float gamma,
                             const float alpha) {
    if ((logits.device().type() != c10::kCUDA) || (labels.device().type() != c10::kCUDA)) {
        AT_ERROR("this focal loss only support gpu mode\n");
    } 
    at::DeviceGuard guard(logits.device());
    return FocalLoss_forward_cuda(logits, labels, gamma, alpha);
}

at::Tensor FocalLoss_backward(const at::Tensor &grad,
                             const at::Tensor &logits,
                             const at::Tensor &labels,
                             const float gamma,
                             const float alpha) {
    // TODO: try AT_ASSERTM
    if ((logits.device().type() != c10::kCUDA) || (labels.device().type() != c10::kCUDA)) {
        AT_ERROR("this focal loss only support gpu mode\n");
    } 
    at::DeviceGuard guard(logits.device());
    return FocalLoss_backward_cuda(grad, logits, labels, gamma, alpha);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("focalloss_forward", &FocalLoss_forward, "focal loss forward");
    m.def("focalloss_backward", &FocalLoss_backward, "focal loss backward");
}
